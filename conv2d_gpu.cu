#include <fstream>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <string>
#include <sstream>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <ctime>
#include <hip/hip_runtime_api.h>

using namespace std;

struct  pixel //to store RGB values
{
	unsigned char r;
	unsigned char g;
	unsigned char b;
};

static void HandleError( hipError_t err, const char *file, int line ) {
	if (err != hipSuccess) {
		cout<<hipGetErrorString(err)<<" in "<< file <<" at line "<< line<<endl;
	}
}


#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__device__ void padding(pixel* Pixel_val, int x_coord, int y_coord, int img_width, int img_height, pixel Px) //padding the image,depending on pixel coordinates, can be replaced by reflect for better result //currently zero padding
{	Px.r=0; Px.g=0; Px.b=0;
	if(x_coord< img_width && y_coord <img_height && x_coord>=0 && y_coord>=0)
		Px=Pixel_val[y_coord*img_width+x_coord];
}


__global__ void vertical_conv(pixel* Pixel_in, pixel* Pixel_out,int img_wd, int img_ht, float* kernel, int k)
{
	float tmp_r, tmp_g, tmp_b;
	int pix_idx=blockIdx.x*blockDim.x + threadIdx.x;
	int row=(int)(pix_idx/img_wd);
	int col=pix_idx%img_wd;
				
	if(row<img_ht && col<img_wd){
		tmp_r=0, tmp_g=0, tmp_b=0;
		for(int l=0;l<k;l++)
		{//doing by 1 D arrays	
			pixel pix_val;
			padding(Pixel_in, col, (row+l-(k-1)/2), img_wd, img_ht, pix_val);
			tmp_r+=pix_val.r * kernel[l];
			tmp_b+=pix_val.b * kernel[l];
			tmp_g+=pix_val.g * kernel[l];
		}

		Pixel_out[pix_idx].r=tmp_r;
		Pixel_out[pix_idx].g=tmp_g;
		Pixel_out[pix_idx].b=tmp_b;
	}
}			


__global__ void horizontal_conv(pixel* Pixel_in, pixel* Pixel_out, int img_wd, int img_ht, float* kernel, int k)
{
	float tmp_r, tmp_b, tmp_g;
	//horizontal convolution
	int pix_idx=blockIdx.x*blockDim.x + threadIdx.x;
	int row=(int)(pix_idx/img_wd);
	int col=pix_idx%img_wd;
	tmp_r=0, tmp_g=0, tmp_b=0;
	if(row<img_ht && col<img_wd)
	{
		for(int l=0; l<k;l++)
		{
			pixel pix_val;
			padding(Pixel_in, col+ l-(k-1)/2, row, img_wd, img_ht, pix_val);
			tmp_r+=pix_val.r * kernel[l];
			tmp_g+=pix_val.g * kernel[l];
			tmp_b+=pix_val.b * kernel[l];
		}
		Pixel_out[pix_idx].r=tmp_r;
		Pixel_out[pix_idx].g=tmp_g;
		Pixel_out[pix_idx].b=tmp_b;
	}
}

int main(int argc, char* argv[])
{
	int nDevices;
	HANDLE_ERROR(hipGetDeviceCount(&nDevices));
	cout<<"number of devices="<<nDevices<<endl;

	for(int i=0;i<nDevices;i++){
	hipDeviceProp_t prop;
    HANDLE_ERROR(hipGetDeviceProperties(&prop, i));
    /*printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    printf("  Memory Clock Rate (KHz): %d\n",
           prop.memoryClockRate);
    printf("  Memory Bus Width (bits): %d\n",
           prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    cout<<"  Total global memory :"<<prop.totalGlobalMem<<endl;
    cout<<"  Shared memory per block :"<<prop.sharedMemPerBlock<<endl;
    cout<<"  Regs per block :"<<prop.regsPerBlock<<endl;
    cout<<"  Warp size :"<<prop.warpSize<<endl;
    cout<<"  Max threads per block :"<<prop.maxThreadsPerBlock<<endl;
    cout<<"  Max threads dimension :"<<prop.maxThreadsDim[0]<<" "<<prop.maxThreadsDim[1]<<" "<<prop.maxThreadsDim[2]<<endl;
    cout<<"  Max grid size: "<<prop.maxGridSize[0]<<" "<<prop.maxThreadsDim[1]<<" "<<prop.maxThreadsDim[2]<<endl;
    */  
  }
	
	time_t start_of_code=time(NULL);
	if(argc != 3) //there should be three arguments
	return 1; //exit and return an error
	
	float sigma = atof(argv[2]); //standard deviation for the gaussian 
    
	//Getting the kernel
	int k=floor(6*sigma);//sigma might have fractional part

	if(k%2==0) k++; //to make the size odd

	float *kernel0 = (float *)malloc(k * sizeof(float)); //y based gaussian
	float *kernel1 = (float *)malloc(k * sizeof(float));	//x based gaussian

	
	float constant1=sqrt(2*M_PI*sigma*sigma);//constants needed to define the kernel
	float constant2=2*sigma*sigma;

	int mid=floor(k/2);
	kernel0[mid]=1/constant1;
	kernel1[mid]=1/constant1;

	for(int i=0;i<floor(k/2);i++)	//using symmetry from center, to generate the separable kernels 
	{
		kernel0[i]=((exp(-(floor(k/2)-i)*(floor(k/2)-i)/constant2)))/constant1;

		kernel1[i]=kernel0[i];

		kernel0[k-1-i]=kernel0[i];

		kernel1[k-1-i]=kernel1[i];

	}

	time_t kernel_generation=time(NULL); //find time taken for kernel generation
	cout<<" Kernel generation time:"<<double(kernel_generation - start_of_code)<<" sec"<<endl;

	
	//reading the PPM file line by line
	ifstream infile;
	infile.open(argv[1]);
	string line;

	int img_wd, img_ht;
	int max_val;
	int line_count=0;

	//line one contains P6, line 2 mentions about gimp version, line 3 stores the height and width
	getline(infile, line);
	istringstream iss1(line);

	//reading first line to check format
	int word;
	string str1;

	iss1>>str1;
	if(str1.compare("P6")!=0)	//comparing magic number
	{
		cout<<"wrong file format"<<endl;
		return 1;
	}

	getline(infile,line); //this line has version related comment, hence ignoring

	getline(infile,line); //this stores image dims
	istringstream iss2(line);
	iss2>>word;// this will be image width
	img_wd=word;
	iss2>>word;// this will be image height
	img_ht=word;

	pixel *Pixel_out=(pixel*)malloc(img_ht*img_wd*sizeof(pixel));
	//storing the pixels as lexicographically
	pixel *Pixel = (pixel*)malloc((img_ht)*(img_wd)*sizeof(pixel));
	//pixel **Pixel_tmp = (pixel **)malloc((img_ht) * sizeof(pixel*)); 
	
	/*for(int i=0;i<(img_ht);i++)
	{
		Pixel_tmp[i]=(pixel*)malloc(img_wd*sizeof(pixel));
		Pixel[i]=(pixel*)malloc((img_wd)*sizeof(pixel));
	}*/



	int pix_cnt=0, cnt=0, row,col;

	getline(infile,line); //this stores max value
	
	istringstream iss3(line);
	iss3>>word;
	max_val=word;//max pixel value

	unsigned int val;

	while (getline(infile, line))
	{
		istringstream iss4(line);
		for (int i=0; i<=line.length();i++)
		{
			if(pix_cnt<img_ht*img_wd)
			{	
				val =((int)line[i]);
				//row=floor(pix_cnt/img_wd);
				//col=pix_cnt%img_wd;
				
				if(cnt%3==0)
				{		
					Pixel[pix_cnt].r=val;
				}
				else if(cnt%3==1)
				{
					Pixel[pix_cnt].g=val;
				}
				else
				{
					Pixel[pix_cnt].b=val;
					pix_cnt++;
				}
				cnt++;
			}
		} 	
		line_count++;		
	}

	time_t reading_file=time(NULL);
	cout<<" File reading time:"<<double(reading_file - kernel_generation)<<" sec"<<endl;


	hipDeviceProp_t prop;
   	HANDLE_ERROR(hipGetDeviceProperties(&prop, 0));


    	int thread_block=prop.maxThreadsPerBlock;
	dim3 DimGrid(ceil(img_ht*img_wd/thread_block),1,1);
	dim3 DimBlock(thread_block,1,1);

	//allocating gpu memory


	pixel *Pixel_tmp_gpu;

	pixel *Pixel_gpu, *Pixel_gpu_res;
   
	HANDLE_ERROR(hipMalloc(&Pixel_gpu_res,img_wd*img_ht*sizeof(pixel))); //allocate space for image on device
	HANDLE_ERROR(hipMalloc(&Pixel_tmp_gpu,img_wd*img_ht*sizeof(pixel))); //allocate space for conv output
	HANDLE_ERROR(hipMalloc(&Pixel_gpu,img_wd*img_ht*sizeof(pixel))); //allocate space for image on device
	
	float *kernel0_gpu, *kernel1_gpu;
	
	//size_t pitch_k0, pitch_k1;

	HANDLE_ERROR(hipMalloc(&kernel0_gpu, k*sizeof(float)));//allocate 

	HANDLE_ERROR(hipMalloc(&kernel1_gpu, k*sizeof(float)));//allocate 
	
	cout<<"memory allocated"<<endl;

	//copying needed data

	HANDLE_ERROR(hipMemcpy(Pixel_gpu, Pixel, img_wd*img_ht*sizeof(pixel),hipMemcpyHostToDevice));

	HANDLE_ERROR(hipMemcpy(kernel0_gpu, kernel0,k*sizeof(float),hipMemcpyHostToDevice));

	HANDLE_ERROR(hipMemcpy(kernel1_gpu,kernel1,k*sizeof(float),hipMemcpyHostToDevice));

	cout<<"memory transfers done"<<endl;

	vertical_conv<<<DimGrid,DimBlock>>>(Pixel_gpu, Pixel_tmp_gpu,img_wd, img_ht,kernel0_gpu,k);
	time_t vertical_convolution=time(NULL);

	cout<<" vertical_convolution time: "<<double(vertical_convolution - reading_file)<<"sec"<<endl;

	
	horizontal_conv<<<DimGrid,DimBlock>>>(Pixel_tmp_gpu, Pixel_gpu_res, img_wd, img_ht, kernel1_gpu, k);
	time_t horizontal_convolution=time(NULL);

	cout<<" horizontal convolution time:" <<double(horizontal_convolution-vertical_convolution)<<" sec"<<endl;


	HANDLE_ERROR(hipMemcpy(Pixel_out,Pixel_gpu_res,img_wd*img_ht*sizeof(pixel),hipMemcpyDeviceToHost));

	//writing this to PPM file
	ofstream ofs;
	ofs.open("output_gpu.ppm", ofstream::out);
	ofs<<"P6\n"<<img_wd<<" "<<img_ht<<"\n"<<max_val<<"\n";
	
	for(int i=0; i <img_ht*img_wd;i++)
	{
		ofs<<Pixel_out[i].r<<Pixel_out[i].g<<Pixel_out[i].b;	//write as ascii
		//cout<<"did we write?";
	}
	
	
	ofs.close();
	time_t end=time(NULL);
	cout<<" Saving the result:"<<double(end-horizontal_convolution)<<" sec"<<endl;

	//display time taken for different processes
	cout<<" Total execution time: "<<double(end-start_of_code)<<" sec"<<endl;

	return 0;
}
